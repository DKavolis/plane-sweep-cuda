#include "hip/hip_runtime.h"
/* More info http://gpu4vision.icg.tugraz.at/papers/2012/graber_master.pdf#pub68
 * section 2.3.3 "Total Generalised Varation Multiview Stereo"
 *
 * Kernels list:
 * update p - TGV2_updateP_kernel()
 * update q - TGV2_updateQ_kernel()
 * update all u - TGV2_updateU_kernel()
 * update r - TGV2_updateR_kernel()
 * calculate It:
 *      transform coordinates at u0- TGV2_transform_coordinates_kernel()
 *      interpolate - bilinear_interpolation_kernel() in kernels.cu
 *      subtract - subtract_kernel()
 * calculate Iu:
 *      transform coordinates at u - TGV2_transform_coordinates_kernel()
 *      interpolate - bilinear_interpolation_kernel() in kernels.cu
 *      calculate coordinate derivatives at u0 - TGV2_calculate_coordinate_derivatives_kernel() (only needed once for each source view)
 *      calculate f(x,u) derivative - TGV2_calculate_derivativeF_kernel()
 *      calculate Iu - TGV2_calculate_Iu_kernel()
 *
 * p, q and u1 are initialised to 0
 * u requires an initial solution
 *
 * p and u1 are rank 2
 * q is rank 4
 * u and r are rank 1
 */

#include <kernels.cu.h>
#include <helper_structs.h>

__global__ void TGV2_updateP_kernel(float * __restrict__ d_Px, float * __restrict__ d_Py,
                                    const float * d_u, const float * __restrict__ d_u1x, const float * __restrict__ d_u1y,
                                    const float alpha1, const float sigma, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        int xn = fminf(ind_x + 1, width - 1);
        int yn = fminf(ind_y + 1, height - 1);

        // p(n+1) = project(p(n) + sigma*alpha1*(grad(ubar(n)) - u1bar(n)))
        // where project(x) = x / max(1, |x|) and x is a vector
        double dx = d_Px[i] + alpha1 * sigma * (d_u[ind_y * width + xn] - d_u[i] - d_u1x[i]);
        double dy = d_Py[i] + alpha1 * sigma * (d_u[yn * width + ind_x] - d_u[i] - d_u1y[i]);
        double d = fmaxf(1.f, sqrt(dx * dx + dy * dy));
        d_Px[i] = dx / d;
        d_Py[i] = dy / d;
    }
}

__global__ void TGV2_updateQ_kernel(float * __restrict__ d_Qx, float * __restrict__ d_Qy,
                                    float * __restrict__ d_Qz, float * __restrict__ d_Qw,
                                    const float * d_u1x, const float * d_u1y,
                                    const float alpha0, const float sigma,
                                    const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        int xn = fminf(ind_x + 1, width - 1);
        int yn = fminf(ind_y + 1, height - 1);

        // q(n+1) = project(q(n) + alpha0*sigma*grad(u1bar(n)))
        // where project(x) = x / max(1, |x|) and x is a vector
        float dx_u1x = d_u1x[ind_y * width + xn] - d_u1x[i];
        float dy_u1x = d_u1x[yn * width + ind_x] - d_u1x[i];
        float dx_u1y = d_u1y[ind_y * width + xn] - d_u1y[i];
        float dy_u1y = d_u1y[yn * width + ind_x] - d_u1y[i];
        double dx = d_Qx[i] + alpha0 * sigma * dx_u1x;
        double dy = d_Qy[i] + alpha0 * sigma * dy_u1y;
        double dz = d_Qz[i] + alpha0 * sigma * (dy_u1x + dx_u1y)/2.0f;
        double dw = d_Qw[i] + alpha0 * sigma * (dy_u1x + dx_u1y)/2.0f;
        double d = fmaxf(1.f, sqrt(dx * dx + dy * dy + dz * dz + dw * dw));
        d_Qx[i] = dx / d;
        d_Qy[i] = dy / d;
        d_Qz[i] = dz / d;
        d_Qw[i] = dw / d;
    }
}

__global__ void TGV2_updateR_kernel(float * __restrict__ d_r, float * __restrict__ d_prodsum,
                                    const float * __restrict__ d_u, const float * __restrict__ d_u0,
                                    const float * __restrict__ d_It, const float * __restrict__ d_Iu,
                                    const float sigma, const float lambda, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        // r(n+1) = project(r(n) + sigma*lambda*(It + (u-u0)*Iu))
        // where project(x) = x / max(1, |x|) and x is a vector
        d_r[i] = d_r[i] + sigma * lambda * (d_It[i] + (d_u[i] - d_u0[i]) * d_Iu[i]);
        d_r[i] = d_r[i] / fmaxf(1.f, fabs(d_r[i]));

        d_prodsum[i] += d_r[i] * d_Iu[i];
    }
}

__global__ void TGV2_updateU_kernel(float * __restrict__ d_u, float * __restrict__ d_u1x, float * __restrict__ d_u1y,
                                    float * __restrict__ d_ubar, float * __restrict__ d_u1xbar, float * __restrict__ d_u1ybar,
                                    const float * d_Px, const float * d_Py, const float * d_Qx, const float * d_Qy,
                                    const float * d_Qz, const float * d_Qw,
                                    const float * __restrict__ d_prodsum,
                                    const float alpha0, const float alpha1, const float tau, const float lambda,
                                    const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        int xp = fmaxf(ind_x - 1, 0.f);
        int yp = fmaxf(ind_y - 1, 0.f);;

        float uprev = d_u[i], u1xprev = d_u1x[i], u1yprev = d_u1y[i];

        // u(n+1) = u(n) - tau*(-alpha1*div(p(n+1)) + lambda*sum_over_i(Iui*ri(n+1)))
        // u1(n+1)= u1(n)- tau*(-alpha1*p(n+1) - alpha0*div(q(n+1)))
        d_u[i] = d_u[i] - tau*(-alpha1 * (d_Px[i] - d_Px[ind_y*width + xp] + d_Py[i] - d_Py[yp*width + ind_x]) + lambda*d_prodsum[i]);
        d_u1x[i] = d_u1x[i] - tau*(-alpha1*d_Px[i] - alpha0*(d_Qx[i] - d_Qx[ind_y*width + xp] + d_Qz[i] - d_Qz[yp*width + ind_x]));
        d_u1y[i] = d_u1y[i] - tau*(-alpha1*d_Py[i] - alpha0*(d_Qz[i] - d_Qz[ind_y*width + xp] + d_Qy[i] - d_Qy[yp*width + ind_x]));

        // ubar(n+1) = 2 * u(n+1) - u(n)
        // u1bar(n+1)= 2 * u1(n+1)- u1(n)
        d_ubar[i] = 2 * d_u[i] - uprev;
        d_u1xbar[i] = 2 * d_u1x[i] - u1xprev;
        d_u1ybar[i] = 2 * d_u1y[i] - u1yprev;
    }
}

// Kernel did not like arrays of known dimensions at compile time, thus each element had to be passed separately...
__global__ void TGV2_transform_coordinates_kernel(float * __restrict__ d_x, float * __restrict__ d_y,
                                                  float * __restrict__ d_X, float * __restrict__ d_Y, float * __restrict__ d_Z,
                                                  const float * __restrict__ d_u,
                                                  const Matrix3D K,
                                                  const Matrix3D Rrel,
                                                  const Vector3D trel,
                                                  const Matrix3D invK,
                                                  const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        // 1 based indexing:
        int x = ind_x + 1;
        int y = ind_y + 1;
        float3 a = make_float3(x, y, 1);

        // Calculate x1 = u * K^(-1) * a
        float3 x1 = d_u[i] * (invK * a);

        // Calculate x2 = [R | t] * x1 = R * x1 + t
        float3 x2 = Rrel * x1 + trel;

        // Store 3D coordinates in the coordinate frame of the 2nd view
        d_X[i] = x2.x;
        d_Y[i] = x2.y;
        d_Z[i] = x2.z;

        // Calculate x1 = K * x2
        x1 = K * x2;

        // Normalize z and revert to 0 based indexing
        d_x[i] = x1.x / x1.z - 1;
        d_y[i] = x1.y / x1.z - 1;
    }
}

__global__ void subtract_kernel(float * __restrict__ d_out, const float * __restrict__ d_in1, const float * __restrict__ d_in2,
                                const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        d_out[i] = d_in1[i] - d_in2[i];
    }
}

__global__ void TGV2_calculate_coordinate_derivatives_kernel(float * __restrict__ d_dX, float * __restrict__ d_dY,
                                                             float * __restrict__ d_dZ,
                                                             const Matrix3D invK,
                                                             const Matrix3D Rrel,
                                                             const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        // 1 based indexing:
        int x = ind_x + 1;
        int y = ind_y + 1;

        // Derivatives are given by grad(X) = Rrel * K^(-1) * x
        // Calculate derivatives x1 = Rrel * K^(-1) * x
        float3 x1 = Rrel * invK * make_float3(x, y, 1);

        // Calculate derivatives
        d_dX[i] = x1.x;
        d_dY[i] = x1.y;
        d_dZ[i] = x1.z;
    }
}

__global__ void TGV2_calculate_derivativeF_kernel(float * __restrict__ d_dfx, float * __restrict__ d_dfy,
                                                  const float * __restrict__ d_X, const float * __restrict__ d_dX,
                                                  const float * __restrict__ d_Y, const float * __restrict__ d_dY,
                                                  const float * __restrict__ d_Z, const float * __restrict__ d_dZ,
                                                  const float fx, const float fy, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        d_dfx[i] = fx * (d_dX[i] * d_Z[i] - d_X[i] * d_dZ[i]) / (d_Z[i] * d_Z[i]);
        d_dfy[i] = fy * (d_dY[i] * d_Z[i] - d_Y[i] * d_dZ[i]) / (d_Z[i] * d_Z[i]);
    }
}

__global__ void TGV2_calculate_Iu_kernel(float * __restrict__ d_Iu, const float * d_I,
                                         const float * __restrict__ d_dfx, const float * __restrict__ d_dfy,
                                         const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        int xn = fminf(ind_x + 1, width -1);
        int yn = fminf(ind_y + 1, height - 1);

        double dx = d_I[ind_y*width + xn] - d_I[i];
        double dy = d_I[yn*width + ind_x] - d_I[i];
        d_Iu[i] = dx * d_dfx[i] + dy * d_dfy[i];
    }
}

__global__ void Anisotropic_diffusion_tensor_kernel(float * __restrict__ d_T11, float * __restrict__ d_T12,
                                                    float * __restrict__ d_T21, float * __restrict__ d_T22,
                                                    const float * __restrict__ d_Img,
                                                    const float beta, const float gamma, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;
        int xn = fminf(ind_x + 1, width -1);
        int yn = fminf(ind_y + 1, height - 1);

        // Calculate image gradient:
        float x = d_Img[ind_y*width+xn] - d_Img[i];
        float y = d_Img[yn*width+ind_x] - d_Img[i];

        // normalize
        float d = sqrt(x * x + y * y);
        float k;

        // Calculate tensor = exp(-beta*|grad(Img)|^gamma)n*trans(n) + m*trans(m),
        // where n is normalized image gradient vector and m is normal to n
        // n = [x y]' and m = [-y x]'
        if (d > 0.f) { // check for division by 0, this avoids QNAN values in tensor
            x = x / d;
            y = y / d;
            k = expf(- beta * powf(d, gamma));
            d_T11[i] = k * x * x + y * y;
            d_T12[i] = (k - 1) * x * y;
            d_T21[i] = (k - 1) * x * y;
            d_T22[i] = k * y * y + x * x;
        }
        else { // set to identity matrix
            d_T11[i] = 1.f;
            d_T12[i] = 0.f;
            d_T21[i] = 0.f;
            d_T22[i] = 1.f;
        }

    }
}

__global__ void TGV2_updateU_tensor_weighed_kernel(float * __restrict__ d_u, float * __restrict__ d_u1x, float * __restrict__ d_u1y,
                                                   const float * __restrict__ d_T11, const float * __restrict__ d_T12,
                                                   const float * __restrict__ d_T21, const float * __restrict__ d_T22,
                                                   float * __restrict__ d_ubar, float * __restrict__ d_u1xbar, float * __restrict__ d_u1ybar,
                                                   const float * d_Px, const float * d_Py, const float * d_Qx, const float * d_Qy,
                                                   const float * d_Qz, const float * d_Qw,
                                                   const float * __restrict__ d_prodsum,
                                                   const float alpha0, const float alpha1, const float tau, const float lambda,
                                                   const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        int xp = fmaxf(ind_x - 1, 0.f);
        int yp = fmaxf(ind_y - 1, 0.f);;

        float uprev = d_u[i], u1xprev = d_u1x[i], u1yprev = d_u1y[i];

        // u(n+1) = u(n) - tau*(-alpha1*div(tensor*p(n+1)) + lambda*sum_over_i(Iui*ri(n+1)))
        // u1(n+1)= u1(n)- tau*(-alpha1*tensor*p(n+1) - alpha0*div(q(n+1)))
        float c_px = d_Px[i], c_py = d_Py[i],
                xp_px = d_Px[ind_y*width+xp], yp_px = d_Px[yp*width+ind_x],
                xp_py = d_Py[ind_y*width+xp], yp_py = d_Py[yp*width+ind_x];

        d_u[i] = d_u[i] - tau*(-alpha1 * (d_T11[i] * (c_px - xp_px) + d_T12[i] * (c_py - xp_py) +
                                          d_T21[i] * (c_px - yp_px) + d_T22[i] * (c_py - yp_py)) + lambda*d_prodsum[i]);
        d_u1x[i] = d_u1x[i] - tau*(-alpha1*(d_T11[i]*c_px+d_T12[i]*c_py) - alpha0*(d_Qx[i] - d_Qx[ind_y*width + xp] + d_Qz[i] - d_Qz[yp*width + ind_x]));
        d_u1y[i] = d_u1y[i] - tau*(-alpha1*(d_T21[i]*c_px+d_T22[i]*c_py) - alpha0*(d_Qz[i] - d_Qz[ind_y*width + xp] + d_Qy[i] - d_Qy[yp*width + ind_x]));

        // ubar(n+1) = 2 * u(n+1) - u(n)
        // u1bar(n+1)= 2 * u1(n+1)- u1(n)
        d_ubar[i] = 2 * d_u[i] - uprev;
        d_u1xbar[i] = 2 * d_u1x[i] - u1xprev;
        d_u1ybar[i] = 2 * d_u1y[i] - u1yprev;
    }
}

__global__ void TGV2_updateP_tensor_weighed_kernel(float * __restrict__ d_Px, float * __restrict__ d_Py,
                                                   const float * __restrict__ d_T11, const float * __restrict__ d_T12,
                                                   const float * __restrict__ d_T21, const float * __restrict__ d_T22,
                                                   const float * d_u, const float * __restrict__ d_u1x, const float * __restrict__ d_u1y,
                                                   const float alpha1, const float sigma, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        int xn = fminf(ind_x + 1, width - 1);
        int yn = fminf(ind_y + 1, height - 1);

        // p(n+1) = project(p(n) + sigma*alpha1*(grad(ubar(n)) - u1bar(n)))
        // where project(x) = x / max(1, |x|) and x is a vector
        double x = d_u[ind_y * width + xn] - d_u[i] - d_u1x[i];
        double y = d_u[yn * width + ind_x] - d_u[i] - d_u1y[i];
        double dx = d_Px[i] + alpha1 * sigma * (d_T11[i] * x + d_T12[i] * y);
        double dy = d_Py[i] + alpha1 * sigma * (d_T21[i] * x + d_T22[i] * y);
        double d = fmaxf(1.f, sqrt(dx * dx + dy * dy));
        d_Px[i] = dx / d;
        d_Py[i] = dy / d;
    }
}

__global__ void TGV2_updateU_sparseDepth_kernel(float * __restrict__ d_u, float * __restrict__ d_u1x, float * __restrict__ d_u1y,
                                                float * __restrict__ d_ubar, float * __restrict__ d_u1xbar, float * __restrict__ d_u1ybar,
                                                const float * __restrict__ d_Px, const float * __restrict__ d_Py,
                                                const float * __restrict__ d_Qx, const float * __restrict__ d_Qy,
                                                const float * __restrict__ d_Qz, const float * __restrict__ d_Qw,
                                                const float * __restrict__ d_w, const float * __restrict__ d_Ds, const float alpha0,
                                                const float alpha1, const float tau, const float theta,
                                                const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        int xp = fmaxf(ind_x - 1, 0.f);
        int yp = fmaxf(ind_y - 1, 0.f);

        float c_px = d_Px[i], c_py = d_Py[i],
                xp_px = d_Px[ind_y*width+xp],
                yp_py = d_Py[yp*width+ind_x];

        d_u[i] = (d_u[i] + tau*(alpha1 * ( (c_px - xp_px) + (c_py - yp_py)) + d_w[i] * d_Ds[i])) / (1 + tau * d_w[i]);
        d_u1x[i] = d_u1x[i] - tau*(-alpha1*(c_px) - alpha0*(d_Qx[i] - d_Qx[ind_y*width + xp] + d_Qz[i] - d_Qz[yp*width + ind_x]));
        d_u1y[i] = d_u1y[i] - tau*(-alpha1*(c_py) - alpha0*(d_Qz[i] - d_Qz[ind_y*width + xp] + d_Qy[i] - d_Qy[yp*width + ind_x]));

        d_ubar[i] = d_u[i] + theta * (d_u[i] - d_ubar[i]);
        d_u1xbar[i] = d_u1x[i] + theta * (d_u1x[i] - d_u1xbar[i]);
        d_u1ybar[i] = d_u1y[i] + theta * (d_u1y[i] - d_u1ybar[i]);
    }
}

__global__ void TGV2_updateU_sparseDepthTensor_kernel(float * __restrict__ d_u, float * __restrict__ d_u1x, float * __restrict__ d_u1y,
                                                      float * __restrict__ d_ubar, float * __restrict__ d_u1xbar, float * __restrict__ d_u1ybar,
                                                      const float * __restrict__ d_T11, const float * __restrict__ d_T12,
                                                      const float * __restrict__ d_T21, const float * __restrict__ d_T22,
                                                      const float * __restrict__ d_Px, const float * __restrict__ d_Py,
                                                      const float * __restrict__ d_Qx, const float * __restrict__ d_Qy,
                                                      const float * __restrict__ d_Qz, const float * __restrict__ d_Qw,
                                                      const float * __restrict__ d_w, const float * __restrict__ d_Ds, const float alpha0,
                                                      const float alpha1, const float tau, const float theta,
                                                      const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        int xp = fmaxf(ind_x - 1, 0.f);
        int yp = fmaxf(ind_y - 1, 0.f);

        float c_px = d_Px[i], c_py = d_Py[i],
                xp_px = d_Px[ind_y*width+xp], yp_px = d_Px[yp*width+ind_x],
                xp_py = d_Py[ind_y*width+xp], yp_py = d_Py[yp*width+ind_x];

        d_u[i] = (d_u[i] + tau*(alpha1 * (d_T11[i] * (c_px - xp_px) + d_T12[i] * (c_py - xp_py) +
                                          d_T21[i] * (c_px - yp_px) + d_T22[i] * (c_py - yp_py)) + d_w[i] * d_Ds[i])) / (1 + tau * d_w[i]);
        d_u1x[i] = d_u1x[i] - tau*(-alpha1*(d_T11[i]*c_px+d_T12[i]*c_py) - alpha0*(d_Qx[i] - d_Qx[ind_y*width + xp] + d_Qz[i] - d_Qz[yp*width + ind_x]));
        d_u1y[i] = d_u1y[i] - tau*(-alpha1*(d_T21[i]*c_px+d_T22[i]*c_py) - alpha0*(d_Qz[i] - d_Qz[ind_y*width + xp] + d_Qy[i] - d_Qy[yp*width + ind_x]));

        d_ubar[i] = d_u[i] + theta * (d_u[i] - d_ubar[i]);
        d_u1xbar[i] = d_u1x[i] + theta * (d_u1x[i] - d_u1xbar[i]);
        d_u1ybar[i] = d_u1y[i] + theta * (d_u1y[i] - d_u1ybar[i]);
    }
}

__global__ void calculateWeights_sparseDepth_kernel(float * __restrict__ d_w, const float * __restrict__ d_Ds, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        if (d_Ds[i] > 0) d_w[i] = 1.f;
        else d_w[i] = 0.f;
    }
}

void TGV2_updateP(float * d_Px, float * d_Py, const float * d_u, const float * d_u1x, const float * d_u1y,
                  const float alpha1, const float sigma, const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_updateP_kernel<<<blocks, threads>>>(d_Px, d_Py, d_u, d_u1x, d_u1y, alpha1, sigma, width, height);
}

void TGV2_updateQ(float * d_Qx, float * d_Qy, float * d_Qz, float * d_Qw, const float * d_u1x, const float * d_u1y,
                  const float alpha0, const float sigma, const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_updateQ_kernel<<<blocks, threads>>>(d_Qx, d_Qy, d_Qz, d_Qw, d_u1x, d_u1y, alpha0, sigma, width, height);
}

void TGV2_updateR(float * d_r, float * d_prodsum, const float * d_u, const float * d_u0, const float * d_It, const float * d_Iu,
                  const float sigma, const float lambda, const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_updateR_kernel<<<blocks, threads>>>(d_r, d_prodsum, d_u, d_u0, d_It, d_Iu, sigma, lambda, width, height);
}

void TGV2_updateU(float * d_u, float * d_u1x, float * d_u1y, float * d_ubar, float * d_u1xbar, float * d_u1ybar,
                  const float * d_Px, const float * d_Py, const float * d_Qx, const float * d_Qy,
                  const float * d_Qz, const float * d_Qw, const float * d_prodsum, const float alpha0,
                  const float alpha1, const float tau, const float lambda,
                  const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_updateU_kernel<<<blocks, threads>>>(d_u, d_u1x, d_u1y, d_ubar, d_u1xbar, d_u1ybar, d_Px, d_Py, d_Qx, d_Qy,
                                             d_Qz, d_Qw, d_prodsum, alpha0, alpha1, tau, lambda, width, height);
}

void TGV2_transform_coordinates(float * d_x, float * d_y, float * d_X, float * d_Y, float * d_Z, const float * d_u,
                                const Matrix3D K, const Matrix3D Rrel, const Vector3D trel, const Matrix3D invK,
const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_transform_coordinates_kernel<<<blocks, threads>>>(d_x, d_y, d_X, d_Y, d_Z, d_u,
                                                           K, Rrel, trel, invK, width, height);
}

void subtract(float * d_out, const float * d_in1, const float * d_in2, const int width, const int height, dim3 blocks, dim3 threads)
{
    subtract_kernel<<<blocks, threads>>>(d_out, d_in1, d_in2, width, height);
}

void TGV2_calculate_coordinate_derivatives(float * d_dX, float * d_dY, float * d_dZ, const Matrix3D invK, const Matrix3D Rrel,
const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_calculate_coordinate_derivatives_kernel<<<blocks, threads>>>(d_dX, d_dY, d_dZ,
                                                                      invK, Rrel, width, height);
}

void TGV2_calculate_derivativeF(float * d_dfx, float * d_dfy, const float * d_X, const float * d_dX, const float * d_Y, const float * d_dY,
                                const float * d_Z, const float * d_dZ, const float fx, const float fy,
                                const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_calculate_derivativeF_kernel<<<blocks, threads>>>(d_dfx, d_dfy, d_X, d_dX, d_Y, d_dY, d_Z, d_dZ, fx, fy, width, height);
}

void TGV2_calculate_Iu(float * d_Iu, const float * d_I, const float * d_dfx, const float * d_dfy,
                       const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_calculate_Iu_kernel<<<blocks, threads>>>(d_Iu, d_I, d_dfx, d_dfy, width, height);
}

void Anisotropic_diffusion_tensor(float * d_T11, float * d_T12, float * d_T21, float * d_T22, const float * d_Img,
                                  const float beta, const float gamma, const int width, const int height,
                                  dim3 blocks, dim3 threads)
{
    Anisotropic_diffusion_tensor_kernel<<<blocks, threads>>>(d_T11, d_T12, d_T21, d_T22, d_Img, beta, gamma, width, height);
}

void TGV2_updateU_tensor_weighed(float * d_u, float * d_u1x, float * d_u1y, const float * d_T11, const float * d_T12,
                                 const float * d_T21, const float * d_T22, float * d_ubar, float * d_u1xbar, float * d_u1ybar,
                                 const float * d_Px, const float * d_Py, const float * d_Qx, const float * d_Qy,
                                 const float * d_Qz, const float * d_Qw, const float * d_prodsum,
                                 const float alpha0, const float alpha1, const float tau, const float lambda,
                                 const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_updateU_tensor_weighed_kernel<<<blocks, threads>>>(d_u, d_u1x, d_u1y, d_T11, d_T12, d_T21, d_T22, d_ubar, d_u1xbar, d_u1ybar,
                                                            d_Px, d_Py, d_Qx, d_Qy, d_Qz, d_Qw, d_prodsum, alpha0, alpha1, tau, lambda,
                                                            width, height);
}

void TGV2_updateP_tensor_weighed(float * d_Px, float * d_Py, const float * d_T11, const float * d_T12, const float * d_T21, const float * d_T22,
                                 const float * d_u, const float * d_u1x, const float * d_u1y, const float alpha1,
                                 const float sigma, const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_updateP_tensor_weighed_kernel<<<blocks, threads>>>(d_Px, d_Py, d_T11, d_T12, d_T21, d_T22, d_u, d_u1x, d_u1y, alpha1,
                                                            sigma, width, height);
}

void TGV2_updateU_sparseDepth(float * d_u, float * d_u1x, float * d_u1y,
                              float * d_ubar, float * d_u1xbar, float * d_u1ybar,
                              const float * d_Px, const float * d_Py,
                              const float * d_Qx, const float * d_Qy,
                              const float * d_Qz, const float * d_Qw,
                              const float * d_w, const float * d_Ds, const float alpha0,
                              const float alpha1, const float tau, const float theta,
                              const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_updateU_sparseDepth_kernel<<<blocks, threads>>>(d_u, d_u1x, d_u1y, d_ubar, d_u1xbar, d_u1ybar, d_Px, d_Py,
                                                         d_Qx, d_Qy, d_Qz, d_Qw, d_w, d_Ds, alpha0, alpha1, tau, theta, width, height);
}

void TGV2_updateU_sparseDepthTensor(float * d_u, float * d_u1x, float * d_u1y,
                                    float * d_ubar, float * d_u1xbar, float * d_u1ybar,
                                    const float * d_T11, const float * d_T12,
                                    const float * d_T21, const float * d_T22,
                                    const float * d_Px, const float * d_Py,
                                    const float * d_Qx, const float * d_Qy,
                                    const float * d_Qz, const float * d_Qw,
                                    const float * d_w, const float * d_Ds, const float alpha0,
                                    const float alpha1, const float tau, const float theta,
                                    const int width, const int height, dim3 blocks, dim3 threads)
{
    TGV2_updateU_sparseDepthTensor_kernel<<<blocks, threads>>>(d_u, d_u1x, d_u1y, d_ubar, d_u1xbar, d_u1ybar, d_T11, d_T12, d_T21, d_T22,
                                                               d_Px, d_Py,
                                                               d_Qx, d_Qy, d_Qz, d_Qw, d_w, d_Ds, alpha0, alpha1, tau, theta, width, height);
}

void calculateWeights_sparseDepth(float * d_w, const float * d_Ds, const int width, const int height, dim3 blocks, dim3 threads)
{
    calculateWeights_sparseDepth_kernel<<<blocks, threads>>>(d_w, d_Ds, width, height);
}
