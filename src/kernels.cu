#include "hip/hip_runtime.h"
#include <kernels.cu.h>
#include <helper_structs.h>

__global__ void bilinear_interpolation_kernel_GPU(float * __restrict__ d_result, const float * __restrict__ d_data,
                                                  const float * __restrict__ d_xout, const float * __restrict__ d_yout,
                                                  const int M1, const int M2, const int N1, const int N2)
{
    const int l = threadIdx.x + blockDim.x * blockIdx.x;
    const int k = threadIdx.y + blockDim.y * blockIdx.y;

    if ((l<N1)&&(k<N2)) {

        float result_temp1, result_temp2;

        const int    ind_x = floor(d_xout[k*N1+l]);
        const float  a     = d_xout[k*N1+l]-ind_x;

        const int    ind_y = floor(d_yout[k*N1+l]);
        const float  b     = d_yout[k*N1+l]-ind_y;

        if ((ind_x < 0) || (ind_y < 0) || (ind_y+1 > M2-1) || (ind_x+1 > M1-1)) { d_result[k*N1+l] = 0.f; return; }

        result_temp1 = a * d_data[ind_y*M1+ind_x+1] + (1 - a) * d_data[ind_y*M1+ind_x];

        result_temp2 = a * d_data[(ind_y+1)*M1+ind_x+1] + (1 - a) * d_data[(ind_y+1)*M1+ind_x];

        d_result[k*N1+l] = b * result_temp2 + (1 - b) * result_temp1;
    }
}

__global__ void transform_indexes_kernel(float * __restrict__ d_x, float * __restrict__ d_y,
                                         const Matrix3D h,
                                         const int width, const int height)
{
    const int l = threadIdx.x + blockDim.x * blockIdx.x;
    const int k = threadIdx.y + blockDim.y * blockIdx.y;

    if ((l < width) && (k < height)) {
        float3 x = h * make_float3(l+1, k+1, 1);
        x = x / x.z - 1;
        d_x[width * k + l] = x.x;
        d_y[width * k + l] = x.y;
    }
}

__global__ void calcNCC_kernel(float * __restrict__ d_ncc, const float * __restrict d_prod_mean,
                               const float * __restrict__ d_mean1, const float * __restrict__ d_mean2,
                               const float * __restrict__ d_std1, const float * __restrict__ d_std2,
                               const float stdthresh1, const float stdthresh2,
                               const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;

        // If either STD is below threshold, set NCC to 0
        if ((d_std1[ind] < stdthresh1) || (d_std2[ind] < stdthresh2)) d_ncc[ind] = 0.f;
        else {
            d_ncc[ind] = (d_prod_mean[ind] - d_mean1[ind] * d_mean2[ind]) / (d_std1[ind] * d_std2[ind]);
        }
    }
}

__global__ void update_arrays_kernel(float * __restrict__ d_depthmap, float * __restrict__ d_bestncc,
                                     const float * __restrict__ d_currentncc, const float current_depth,
                                     const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;

        // Update if better correspondance was found
        if (d_currentncc[ind] > d_bestncc[ind]){
            d_bestncc[ind] = d_currentncc[ind];
            d_depthmap[ind] = current_depth;
        }
    }
}

__global__ void sum_depthmap_NCC_kernel(float * __restrict__ d_depthmap_out, float * __restrict__ d_count,
                                        const float * __restrict__ d_depthmap, const float * __restrict__ d_ncc,
                                        const float nccthreshold,
                                        const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;

        // Sum if NCC is above threshold
        if (d_ncc[ind] > nccthreshold){
            d_depthmap_out[ind] += d_depthmap[ind];
            d_count[ind]++;
        }
    }
}

__global__ void calculate_STD_kernel(float * __restrict__ d_std, const float * __restrict__ d_mean,
                                     const float * __restrict__ d_mean_of_squares,
                                     const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;

        // variance (easy but numerically unstable method)
        float var = d_mean_of_squares[ind] - d_mean[ind] * d_mean[ind];

        // check for negative variance
        if (var > 0) d_std[ind] = sqrt(var);
        else d_std[ind] = 0.f;
    }
}

__global__ void set_value_kernel(float * __restrict__ d_output, const float value, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;
        d_output[ind] = value;
    }
}

__global__ void element_multiply_kernel(float * __restrict__ d_output, const float * __restrict__ d_input1,
                                        const float * __restrict__ d_input2,
                                        const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        d_output[ind_x+ind_y*width] = d_input1[ind_x+ind_y*width] * d_input2[ind_x+ind_y*width];
    }
}

__global__ void element_rdivide_kernel(float * __restrict__ d_output, const float * __restrict__ d_input1,
                                       const float * __restrict__ d_input2,
                                       const int width, const int height, const float QNaN)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;
        if (d_input2[ind] != 0) d_output[ind] = d_input1[ind] / d_input2[ind];
        else d_output[ind] = QNaN;
    }
}

__global__ void convert_float_to_uchar_kernel(unsigned char * __restrict__ d_output, const float * __restrict__ d_input,
                                              const float min, const float max,
                                              const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;

        if (max == min) d_output[ind] = (unsigned char)(UCHAR_MAX / 2);
        else {
            if (min > max){
                if (d_input[ind] > min) d_output[ind] = UCHAR_MAX;
                else if (d_input[ind] < max) d_output[ind] = NULL;
                else if (d_input[ind] == d_input[ind]) d_output[ind] = (unsigned char)(UCHAR_MAX * (d_input[ind] - max) / (min - max));
                else d_output[ind] = UCHAR_MAX;
            }
            else {
                if (d_input[ind] > max) d_output[ind] = UCHAR_MAX;
                else if (d_input[ind] < min) d_output[ind] = NULL;
                else if (d_input[ind] == d_input[ind]) d_output[ind] = (unsigned char)(UCHAR_MAX * (d_input[ind] - min) / (max - min));
                else d_output[ind] = UCHAR_MAX;
            }
        }
    }
}

__global__ void windowed_mean_row_kernel(float * __restrict__ d_output, const float * __restrict__ d_input,
                                         const unsigned int winsize, const bool squared,
                                         const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;

        float mean = 0.f;
        int n = winsize / 2;
        int k;

        for (int i = -n; i <= n; i++){
            k = ind_x + i;
            if (k < 0) k = -k;
            if (k > width - 1) k = 2 * (width - 1) - k;
            if (squared) mean += d_input[ind_y * width + k] * d_input[ind_y * width + k];
            else mean += d_input[ind_y * width + k];
        }
        d_output[ind] = mean / (float)winsize;
    }
}

__global__ void windowed_mean_column_kernel(float * __restrict__ d_output, const float * __restrict__ d_input,
                                            const unsigned int winsize, const bool squared,
                                            const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;

        float mean = 0.f;
        int n = winsize / 2;
        int k;

        for (int i = -n; i <= n; i++){
            k = ind_y + i;
            if (k < 0) k = -k;
            if (k > height - 1) k = 2 * (height - 1) - k;
            if (squared) mean += d_input[k * width + ind_x] * d_input[k * width + ind_x];
            else mean += d_input[k * width + ind_x];
        }
        d_output[ind] = mean / (float)winsize;
    }
}

__global__ void convert_uchar_to_float_kernel(float * __restrict__ d_output, const unsigned char * __restrict__ d_input,
                                              const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;

        d_output[ind] = d_input[ind];
    }
}

__global__ void denoising_TVL1_calculateP_kernel(float * __restrict__ d_Px, float * __restrict__ d_Py,
                                                 const float * d_input, const float sigma,
                                                 const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;
        int ny = ind_y + 1;
        if (ny > height - 1) ny = height - 1;

        double dx, dy, m;

        if (ind_x == width - 1){ // last column
            dy = (d_input[ny * width + ind_x] - d_input[ind]) * sigma + d_Py[ind];
            m = 1.f / dy;
            if (m < 0.f) m = -m;
            if (m > 1.f) m = 1.f;
            d_Px[ind] = 0.f;
            d_Py[ind] = dy * m;
        }
        else {
            dx = (d_input[ind + 1] - d_input[ind]) * sigma + d_Px[ind];
            dy = (d_input[ny * width + ind_x] - d_input[ind]) * sigma + d_Py[ind];
            m = 1.f / sqrt(dx * dx + dy * dy);
            if (m > 1.f) m = 1.f;
            d_Px[ind] = dx * m;
            d_Py[ind] = dy * m;
        }
    }
}

__global__ void denoising_TVL1_calculateP_tensor_weighed_kernel(float * __restrict__ d_Px, float * __restrict__ d_Py,
                                                                const float * __restrict__ d_T11, const float * __restrict__ d_T12,
                                                                const float * __restrict__ d_T21, const float * __restrict__ d_T22,
                                                                const float * d_input, const float sigma,
                                                                const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {

        const int i = ind_y * width + ind_x;

        int xn = fminf(ind_x + 1, width - 1);
        int yn = fminf(ind_y + 1, height - 1);

        double x = d_input[ind_y * width + xn] - d_input[i];
        double y = d_input[yn * width + ind_x] - d_input[i];
        double dx = d_Px[i] + sigma * (d_T11[i] * x + d_T12[i] * y);
        double dy = d_Py[i] + sigma * (d_T21[i] * x + d_T22[i] * y);
        double d = fmaxf(1.f, sqrt(dx * dx + dy * dy));
        d_Px[i] = dx / d;
        d_Py[i] = dy / d;
    }
}

__global__ void element_scale_kernel(float * __restrict__ d_output, const float scale, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;
        d_output[ind] = d_output[ind] * scale;
    }
}

__global__ void element_add_kernel(float * __restrict__ d_output, const float value, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;
        d_output[ind] += value;
    }
}

__global__ void set_QNAN_value_kernel(float * __restrict__ d_output, const float value, const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;
        if (d_output[ind] != d_output[ind]) d_output[ind] = value;
    }
}

__global__ void denoising_TVL1_update_kernel(float * __restrict__ d_output, float * __restrict__ d_R,
                                             const float * d_Px, const float * d_Py, const float * __restrict__ d_origin,
                                             const float tau, const float theta, const float lambda, const float sigma,
                                             const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int ind = ind_y * width + ind_x;
        double x_new;
        int yp = ind_y - 1;
        if (yp < 0) yp = 0;

        d_R[ind] += d_origin[ind];
        d_R[ind] += sigma * d_output[ind];
        if (d_R[ind] > lambda) d_R[ind] = lambda;
        if (d_R[ind] < -lambda) d_R[ind] = -lambda;

        if (ind_x == 0){
            x_new = d_output[ind] + tau*(d_Py[ind] - d_Py[yp * width + ind_x]) - tau * d_R[ind];
            d_output[ind] = x_new + theta*(x_new - d_output[ind]);
        }
        else {
            x_new = d_output[ind] + tau*(d_Px[ind] - d_Px[ind - 1] + d_Py[ind] - d_Py[yp * width + ind_x]) - tau * d_R[ind];
            d_output[ind] = x_new + theta*(x_new - d_output[ind]);
        }
    }
}

__global__ void denoising_TVL1_update_tensor_weighed_kernel(float * __restrict__ d_output, float * __restrict__ d_R,
                                                            const float * d_Px, const float * d_Py, const float * __restrict__ d_origin,
                                                            const float * __restrict__ d_T11, const float * __restrict__ d_T12,
                                                            const float * __restrict__ d_T21, const float * __restrict__ d_T22,
                                                            const float tau, const float theta, const float lambda, const float sigma,
                                                            const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {

        const int i = ind_y * width + ind_x;

        int xp = fmaxf(ind_x - 1, 0.f);
        int yp = fmaxf(ind_y - 1, 0.f);

        d_R[i] += d_origin[i];
        d_R[i] += sigma * d_output[i];
        if (d_R[i] > lambda) d_R[i] = lambda;
        if (d_R[i] < -lambda) d_R[i] = -lambda;

        float   c_px = d_Px[i], c_py = d_Py[i],
                xp_px = d_Px[ind_y*width+xp], yp_px = d_Px[yp*width+ind_x],
                xp_py = d_Py[ind_y*width+xp], yp_py = d_Py[yp*width+ind_x];

        double x_new = d_output[i] + tau*((d_T11[i] * (c_px - xp_px) + d_T12[i] * (c_py - xp_py) +
                                           d_T21[i] * (c_px - yp_px) + d_T22[i] * (c_py - yp_py)) - d_R[i]);

        d_output[i] = x_new + theta*(x_new - d_output[i]);
    }
}

__global__ void compute3D_kernel(float * __restrict__ d_x, float * __restrict__ d_y, float * __restrict__ d_z,
                                 const Matrix3D Rrel,
                                 const Vector3D trel,
                                 const Matrix3D invK,
                                 const int width, const int height)
{
    const int ind_x = threadIdx.x + blockDim.x * blockIdx.x;
    const int ind_y = threadIdx.y + blockDim.y * blockIdx.y;

    if ((ind_x < width) && (ind_y < height)) {
        const int i = ind_y * width + ind_x;

        int x = ind_x + 1, y = ind_y + 1;
        float z = d_z[i];

        if (z < 5.5)
        {
            float3 x1 = Rrel * (z * invK * make_float3(x, y, 1)) + trel;
            d_x[i] = x1.x;
            d_y[i] = x1.y;
            d_z[i] = x1.z;
        }
        else d_z[i] = -9.f;
    }
}

void transform_indexes(float * d_x, float *  d_y,
                       const Matrix3D h,
                       const int width, const int height, dim3 blocks, dim3 threads)
{
    transform_indexes_kernel<<<blocks, threads>>>(d_x, d_y,
                                                  h,
                                                  width, height);
}

void bilinear_interpolation(float * d_result, const float * d_data,
                            const float * d_xout, const float * d_yout,
                            const int M1, const int M2, const int N1, const int N2,
                            dim3 blocks, dim3 threads)
{
    bilinear_interpolation_kernel_GPU<<<blocks, threads>>>(d_result, d_data, d_xout, d_yout, M1, M2, N1, N2);
}

void calcNCC(float * d_ncc, const float * d_prod_mean,
             const float * d_mean1, const float * d_mean2,
             const float * d_std1, const float * d_std2,
             const float stdthresh1, const float stdthresh2,
             const int width, const int height,
             dim3 blocks, dim3 threads)
{
    calcNCC_kernel<<<blocks, threads>>>(d_ncc, d_prod_mean,
                                        d_mean1, d_mean2,
                                        d_std1, d_std2,
                                        stdthresh1, stdthresh2,
                                        width, height);
}

void update_arrays(float * d_depthmap, float * d_bestncc,
                   const float * d_currentncc, const float current_depth,
                   const int width, const int height,
                   dim3 blocks, dim3 threads)
{
    update_arrays_kernel<<<blocks, threads>>>(d_depthmap, d_bestncc,
                                              d_currentncc, current_depth,
                                              width, height);
}

void sum_depthmap_NCC(float * d_depthmap_out, float * d_count,
                      const float * d_depthmap, const float * d_ncc,
                      const float nccthreshold,
                      const int width, const int height,
                      dim3 blocks, dim3 threads)
{
    sum_depthmap_NCC_kernel<<<blocks, threads>>>(d_depthmap_out, d_count,
                                                 d_depthmap, d_ncc,
                                                 nccthreshold,
                                                 width, height);
}

void calculate_STD(float * d_std, const float * d_mean,
                   const float * d_mean_of_squares,
                   const int width, const int height,
                   dim3 blocks, dim3 threads)
{
    calculate_STD_kernel<<<blocks, threads>>>(d_std, d_mean,
                                              d_mean_of_squares,
                                              width, height);
}

void set_value(float * d_output, const float value, const int width, const int height, dim3 blocks, dim3 threads)
{
    set_value_kernel<<<blocks, threads>>>(d_output, value, width, height);
}

void element_multiply(float * d_output, const float * d_input1,
                      const float * d_input2,
                      const int width, const int height,
                      dim3 blocks, dim3 threads)
{
    element_multiply_kernel<<<blocks, threads>>>(d_output, d_input1, d_input2,
                                                 width, height);
}

void element_rdivide(float * d_output, const float * d_input1,
                     const float * d_input2,
                     const int width, const int height,
                     dim3 blocks, dim3 threads)
{
    const float QNan = std::numeric_limits<float>::quiet_NaN();
    element_rdivide_kernel<<<blocks, threads>>>(d_output, d_input1, d_input2, width, height, QNan);
}

void convert_float_to_uchar(unsigned char * d_output, const float * d_input,
                            const float min, const float max,
                            const int width, const int height,
                            dim3 blocks, dim3 threads)
{
    convert_float_to_uchar_kernel<<<blocks, threads>>>(d_output, d_input, min, max, width, height);
}

void windowed_mean_row(float * d_output, const float * d_input,
                       const unsigned int winsize, const bool squared,
                       const int width, const int height, dim3 blocks, dim3 threads)
{
    windowed_mean_row_kernel<<<blocks, threads>>>(d_output, d_input, winsize, squared,
                                                  width, height);
}

void windowed_mean_column(float * d_output, const float * d_input,
                          const unsigned int winsize, const bool squared,
                          const int width, const int height, dim3 blocks, dim3 threads)
{
    windowed_mean_column_kernel<<<blocks, threads>>>(d_output, d_input, winsize, squared,
                                                     width, height);
}

void convert_uchar_to_float(float * d_output, const unsigned char * d_input,
                            const int width, const int height, dim3 blocks, dim3 threads)
{
    convert_uchar_to_float_kernel<<<blocks, threads>>>(d_output, d_input, width, height);
}

void denoising_TVL1_calculateP(float * d_Px, float * d_Py,
                               const float * d_input, const float sigma,
                               const int width, const int height,
                               dim3 blocks, dim3 threads)
{
    denoising_TVL1_calculateP_kernel<<<blocks, threads>>>(d_Px, d_Py, d_input, sigma, width, height);
}

void denoising_TVL1_calculateP_tensor_weighed(float * d_Px, float * d_Py,
                                              const float * d_T11, const float * d_T12, const float * d_T21, const float * d_T22,
                                              const float * d_input, const float sigma,
                                              const int width, const int height,
                                              dim3 blocks, dim3 threads)
{
    denoising_TVL1_calculateP_tensor_weighed_kernel<<<blocks, threads>>>(d_Px, d_Py, d_T11, d_T12, d_T21, d_T22,
                                                                         d_input, sigma, width, height);
}

void element_scale(float * d_output, const float scale, const int width, const int height, dim3 blocks, dim3 threads)
{
    element_scale_kernel<<<blocks, threads>>>(d_output, scale, width, height);
}

void element_add(float * d_output, const float value, const int width, const int height, dim3 blocks, dim3 threads)
{
    element_add_kernel<<<blocks, threads>>>(d_output, value, width, height);
}

void set_QNAN_value(float * d_output, const float value, const int width, const int height, dim3 blocks, dim3 threads)
{
    set_QNAN_value_kernel<<<blocks, threads>>>(d_output, value, width, height);
}

void denoising_TVL1_update(float * d_output, float * d_R,
                           const float * d_Px, const float * d_Py, const float * d_origin,
                           const float tau, const float theta, const float lambda, const float sigma,
                           const int width, const int height, dim3 blocks, dim3 threads)
{
    denoising_TVL1_update_kernel<<<blocks, threads>>>(d_output, d_R, d_Px, d_Py, d_origin,
                                                      tau, theta, lambda, sigma, width, height);
}

void denoising_TVL1_update_tensor_weighed(float * d_output, float * d_R,
                                          const float * d_Px, const float * d_Py, const float * d_origin,
                                          const float * d_T11, const float * d_T12, const float * d_T21, const float * d_T22,
                                          const float tau, const float theta, const float lambda, const float sigma,
                                          const int width, const int height, dim3 blocks, dim3 threads)
{
    denoising_TVL1_update_tensor_weighed_kernel<<<blocks, threads>>>(d_output, d_R, d_Px, d_Py, d_origin, d_T11, d_T12, d_T21, d_T22,
                                                                     tau, theta, lambda, sigma, width, height);
}

void compute3D(float * d_x, float * d_y, float * d_z, const Matrix3D Rrel, const Vector3D trel,
               const Matrix3D invK, const int width, const int height, dim3 blocks, dim3 threads)
{
    compute3D_kernel<<<blocks, threads>>>(d_x, d_y, d_z,
                                          Rrel, trel, invK, width, height);
}
